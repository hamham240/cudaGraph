#include "../../cudaGraph/algos/cudaAPSP.hpp"

namespace cudaGraph
{
    std::vector<std::vector<int>> launchAPSP(Graph &g)
    {
        int vertexCount = g.vertices.size();
        int edgeCount = g.edges.size();

        int sizeOfVertices = vertexCount * sizeof(int);
        
        int block = 1024;
        int grid = (vertexCount/ 1024) + 1;

        int* d_cost = NULL;

        checkError(hipMalloc(&d_cost, sizeOfVertices));

        std::vector<std::vector<int>> costs(vertexCount);

        for (int i = 0; i < vertexCount; i++)
        {
            SSSP_initVectors<<<grid, block>>>(d_cost, vertexCount, i);
            hipDeviceSynchronize();
    
            for (int j = 0; j < vertexCount - 1; j++)
            {
                bellmanFord<<<grid, block>>>(g.d_startIndices, g.d_endIndices, g.d_edges, g.d_weights, d_cost, vertexCount);
                hipDeviceSynchronize();
            }
    
            std::vector<int> currentVertexCost(vertexCount);
    
            checkError(hipMemcpy(currentVertexCost.data(), d_cost, sizeOfVertices, hipMemcpyDeviceToHost));

            costs[i] = currentVertexCost;
        }

        checkError(hipFree(d_cost));

        return costs;
    }

    float launchTimedAPSP(Graph &g)
    {
        hipEvent_t start, stop;
        float time;

        int vertexCount = g.vertices.size();
        int edgeCount = g.edges.size();

        int sizeOfVertices = vertexCount * sizeof(int);
        
        int block = 1024;
        int grid = (vertexCount/ 1024) + 1;

        int* d_cost = NULL;

        checkError(hipEventCreate(&start));
        checkError(hipEventCreate(&stop));

        hipEventRecord(start);

        checkError(hipMalloc(&d_cost, sizeOfVertices));

        std::vector<std::vector<int>> costs(vertexCount);

        for (int i = 0; i < vertexCount; i++)
        {
            SSSP_initVectors<<<grid, block>>>(d_cost, vertexCount, i);
            hipDeviceSynchronize();
    
            for (int j = 0; j < vertexCount - 1; j++)
            {
                bellmanFord<<<grid, block>>>(g.d_startIndices, g.d_endIndices, g.d_edges, g.d_weights, d_cost, vertexCount);
                hipDeviceSynchronize();
            }
    
            std::vector<int> currentVertexCost(vertexCount);
    
            checkError(hipMemcpy(currentVertexCost.data(), d_cost, sizeOfVertices, hipMemcpyDeviceToHost));

            costs[i] = currentVertexCost;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        checkError(hipFree(d_cost));

        return time;
    }
}